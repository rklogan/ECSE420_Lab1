#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>
#include <chrono>
#include "lodepng.h"
#include <cstdlib>
using namespace std;

bool TESTMODE = true;

__global__ void maxPool(unsigned char* ip_img, unsigned char* op_img, unsigned int* d_width, unsigned int* d_pix_per_thread, unsigned int* d_op_pixels) {
	int startIndex = threadIdx.x * *d_pix_per_thread;
	int op_width = *d_width / 2;
	printf("%d\n", *d_pix_per_thread);
	int curr_index = startIndex;

	for (int i{ 0 }; i < *d_pix_per_thread; i++) {
		//convert into catesian coods
		int op_coods[2], ip_coods[2];
		op_coods[0] = curr_index % op_width;
		op_coods[1] = curr_index / op_width;

		ip_coods[0] = 2 * op_coods[0];
		ip_coods[1] = 2 * op_coods[1];
			
		//convert back from cartesian to linear
		int ip_idx = 4 * (ip_coods[0] + ip_coods[1] * *d_width);
		
		//search the 2x2 for the max value of each layer
		unsigned char max_rgba[4] = { 0,0,0,0 };
		for (int j{ 0 }; j < 4; j++) {
			int target = ip_idx;

			switch (j)	//switch to visit each pixel in the 2x2
			{
			case 0: break; //we already computed the starting pixel ip_idx
			case 1: target += 4; break;	//one pixel to the right
			case 2: target += 4 * *d_width; break;	//one row below
			case 3: target += 4 * *d_width + 4; break;	//down and right
			default: printf("This should be unreachable....."); break;
			}

			//check if we beat the old maximum on any layer
			for (int k{ 0 }; k < 4; k++) {
				if (ip_img[target + k] > max_rgba[k])
					max_rgba[k] = ip_img[target + k];
			}

		}
		
		//write to output
		for (int j{ 0 }; j < 4; j++)
			op_img[4 * curr_index + j] = max_rgba[j];			//TODO replace RHS
		curr_index++;
		if (curr_index > * d_op_pixels) return;			//overflow protection
	}
}

vector<unsigned char> singleThreadedPool(vector<unsigned char> ip_img, int ip_width, int ip_height) {
	printf("hit");
	vector<unsigned char> output;
	
	int op_width = ip_width / 2;
	int op_height = ip_height / 2;
	
	//loop for each output pixel (in order)
	for (int op_y{ 0 }; op_y < op_height; op_y++) {
		for (int op_x{ 0 }; op_x < op_width; op_x++) {
			//get the cartesion for the input
			int ip_coord[2];
			ip_coord[0] = 2 * op_x;
			ip_coord[1] = 2 * op_y;
			int ip_idx = 4 * (ip_coord[0] + ip_coord[1] * ip_width);

			//search the pixels in the 2x2
			unsigned char max_rgba[4] = { 0,0,0,0 };
			for (int i{ 0 }; i < 4; i++) {
				int target = ip_idx;

				switch (i) {	//switch to visit each pixel
				case 0: break;
				case 1: target += 4; break;
				case 2: target += 4 * ip_width; break;
				case 3: target += 4 * ip_width; break;
				default: printf("This should be unreachable...."); break;
				}

				//check to see if we have a new winner
				for (int j{ 0 }; j < 4; j++) {
					if (ip_img[target + j] > max_rgba[j])
						max_rgba[j] = ip_img[target + j];
				}
			}

			//write to output
			for (int i{ 0 }; i < 4; i++)
				output.push_back(max_rgba[i]);

		}
	}
	return output;

}

int main(int argc, char* argv[]) {
	//default CLAs
	string ip_img_name = "test.png";
	string op_img_name = "output.png";
	int num_threads = 1;

	//get command line args
	if (argc >= 2) ip_img_name = string(argv[1]);
	if (argc >= 3) op_img_name = string(argv[2]);
	if (argc >= 4) num_threads = atoi(argv[3]);

	vector<unsigned char> img;
	unsigned int width, height;

	//load the image
	unsigned error = lodepng::decode(img, width, height, ip_img_name);
	//if (error) cout << "Error loading image: " << error << ": " << lodepng_error_text(error) << endl;

	if (num_threads == 1) {
		auto start = chrono::high_resolution_clock::now();

		vector<unsigned char> output = singleThreadedPool(img, width, height);

		auto end = chrono::high_resolution_clock::now();
	
		error = lodepng::encode(op_img_name, output, width/2, height/2);

		auto time = chrono::duration_cast<chrono::nanoseconds>(end - start).count();
		if (TESTMODE)
			cout << time << endl;
	}
	else{
		int op_width = width / 2;
		int op_height = height / 2;
		int op_pixels = op_width * op_height * 4;
		int op_size = op_pixels * sizeof(char);
		int ip_size = size(img) * sizeof(char);

		int pixels_per_thread = ((op_width * op_height) + 1) / num_threads;

		//begin timer for parallel overhead
		auto begin_overhead = chrono::high_resolution_clock::now();

		//device memory allocation
		unsigned char* d_ip_img;
		unsigned char* d_op_img;
		unsigned int* d_width;
		unsigned int* d_pix_per_thread;
		unsigned int* d_op_pixels;
		hipMalloc((void**)& d_ip_img, ip_size);
		hipMalloc((void**)& d_op_img, op_size);
		hipMalloc((void**)& d_width, sizeof(int));
		hipMalloc((void**)& d_pix_per_thread, sizeof(int));
		hipMalloc((void**)& d_op_pixels, sizeof(int));

		//host memory allocation
		unsigned char* ip_img_ptr = (unsigned char*)malloc(ip_size);
		for (int i{ 0 }; i < size(img); i++)
			ip_img_ptr[i] = img[i];
		unsigned char* op_img_ptr = (unsigned char*)malloc(op_size);

		//copy to device
		hipMemcpy(d_ip_img, ip_img_ptr, ip_size, hipMemcpyHostToDevice);
		hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_pix_per_thread, &pixels_per_thread, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_op_pixels, &op_pixels, sizeof(int), hipMemcpyHostToDevice);

		/******************** Parallel Computations ********************/
		auto being_parallel = chrono::high_resolution_clock::now();
	
		maxPool<<<1, num_threads >>> (d_ip_img, d_op_img, d_width, d_pix_per_thread, d_op_pixels);

		auto end_parallel = chrono::high_resolution_clock::now();
		/************************* End Parallel *************************/

		//copy data from device
		hipMemcpy(op_img_ptr, d_op_img, op_size, hipMemcpyDeviceToHost);

		//free device
		hipFree(d_ip_img); hipFree(d_op_img);
		hipFree(d_width); hipFree(d_pix_per_thread);

		auto end_overhead = chrono::high_resolution_clock::now();

		//convert back into a vector
		vector<unsigned char> op_vector;
		for (int i{ 0 }; i < (op_height * op_width) * 4; i++)
			op_vector.push_back(op_img_ptr[i]);

		error = lodepng::encode(op_img_name, op_vector, op_width, op_height);
		//if (error)
		//	cout << "Encode error: " << error << ": " << lodepng_error_text(error) << endl;

		//cleanup
		free(op_img_ptr); free(ip_img_ptr);

		//display timing data
		auto time_with_overhead = chrono::duration_cast<chrono::nanoseconds>(end_overhead - begin_overhead).count();
		auto time_parallel = chrono::duration_cast<chrono::nanoseconds>(end_parallel - begin_overhead).count();
		if (TESTMODE)
			cout << time_with_overhead << ',' << time_parallel << endl;
	}
	return 0;
}